#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"


#define DYNAMIC_BLOCK_SIZE 0
// tried out an approach which in addition to reducing the # of blocks when the number of threads needed is small would reduce the size of the one remaining block when the amount of threads is less than that needs, but doesn't seem like it particularly benefits. Leaving in as an option but not using

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void kernUpSweep(int n, int dExpo, int* data/*, const int* idata*/) {
            // TODO I assume should invoke these with only the number of blocks actually used rather than with all the blocks when most don't do any, but will do following structure more literally first?
            // TODO probably worth trying out shared memory way--I guess would move loop into here w/ syncs then change external loop to just when needing to cross boundaries
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return; // sounds like this way is generally better (more explicit that thread can stop)
            }
            int d2 = dExpo << 1;
            int k = (index) * d2;
            //if (k >= n) {
                //return; 
            //}
            //if (k < n) {
                //odata[k + d2 - 1] = idata[k + dExpo - 1] + idata[k + d2 - 1];
                // does this need a separate idata odata? I think no others operate on it this step
                data[k + d2 - 1] += data[k + dExpo - 1];
            //}
        }
        __global__ void kernDownSweep(int n, int dExpo, int* data) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
            int d2 = dExpo << 1;
            int k = (index)*d2;

            int t = data[k + dExpo - 1];
            data[k + dExpo - 1] = data[k + d2 - 1];
            data[k + d2 - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO not finished yet, doesn't work yet
            int blockSize = 32; // Seems to perform best here for 32 or 64 but the change is generally pretty minor
            int dTarget = ilog2ceil(n);
            int pow2Size = 1 << dTarget;
            int nCap = pow2Size >> 1;
            dim3 fullBlocksPerGrid((nCap + blockSize - 1) / blockSize);
            // TODO I'm still not sure if getting fullBlocksPerGrid right 100%, might be overshooting
            int* dev_idata;
            int* dev_odata;

            // TODO note this pads to the whole next power of 2, was mentioned but can't recall if they said a way about that?
            // TODO I think want to rewrite into using shared memory way but that's extra credit so I think don't need to
            //   moving on for now but plan to revisit and do that
            //   also not sure if I've already done what part 5 is referring to? since I don't do modulo and such and decrease # of threads

            hipMalloc((void**)&dev_idata, pow2Size * sizeof(int));
            //hipMalloc((void**)&dev_odata, n * sizeof(int));
            
            hipMemset(dev_idata + n, 0, sizeof(int) * (pow2Size - n));
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice); 

            //hipMemset(dev_idata, 0, sizeof(int));
            //hipMemcpy(dev_idata + 1, idata, sizeof(int) * (n - 1), hipMemcpyHostToDevice);

            // TODO make sure to deal with size not 2^x
#if DYNAMIC_BLOCK_SIZE
            int blockSize2 = blockSize;
#endif

            timer().startGpuTimer();

            // TODO
            
            // up-sweep
            int dExpo = 1; // = 2^(d)
            
            for (int d = 0; d < dTarget; ++d) {
#if DYNAMIC_BLOCK_SIZE
                kernUpSweep<<<fullBlocksPerGrid, blockSize2 >>> (nCap, dExpo, dev_idata);
#else
                kernUpSweep<<<fullBlocksPerGrid, blockSize >>> (nCap, dExpo, dev_idata);
#endif
                if (d < dTarget - 1) {
                    //fullBlocksPerGrid.x >>= 1;
                    //fullBlocksPerGrid = dim3((n / dExpo + blockSize - 1) / blockSize);
                    //fullBlocksPerGrid.x = (fullBlocksPerGrid.x - 1) / 2 + 1; // TODO reduce # of blocks accordingly
                    dExpo <<= 1;
                    nCap >>= 1;
                    fullBlocksPerGrid.x = ((nCap + blockSize - 1) / blockSize); // Not sure this is totally the best way to set this but does massively reduce runtime (e.g. ~6ms to ~2ms)
                    //std::swap(dev_idata, dev_odata);

#if DYNAMIC_BLOCK_SIZE
                    // could reduce blockSize when gets really low but not sure worth the effort
                    //  doesn't seem to really matter, so just turning off but making an option anyway
                    if (nCap < blockSize) {
                        blockSize2 = nCap;
                        //blockSize2 = ((nCap - 1) / 32 + 1) * 32;
                    }
                    else {
                        blockSize2 = blockSize;
                    }
#endif
                }
            }
            // down-sweep
            hipMemset(dev_idata + (pow2Size - 1), 0, sizeof(int)); // TODO make sure that's right
            //fullBlocksPerGrid = dim3((pow2Size + blockSize - 1) / blockSize);
            // TODO make sure dExpo right
            for (int d = dTarget - 1; d >= 0; --d) {
#if DYNAMIC_BLOCK_SIZE
                if (nCap < blockSize) {
                    blockSize2 = nCap;
                }
                else {
                    blockSize2 = blockSize;
                }
                kernDownSweep<<<fullBlocksPerGrid, blockSize2>>>(nCap, dExpo, dev_idata);
#else
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(nCap, dExpo, dev_idata);
#endif
                // TODO make fullBlocksPerGrid right;
                //fullBlocksPerGrid.x = (dExpo + blockSize - 1) / blockSize; // TODO make sure set properly but works on super basic case
                if (d > 0) {
                    dExpo >>= 1;
                    nCap <<= 1;
                    fullBlocksPerGrid.x = ((nCap + blockSize - 1) / blockSize);
                }
            }


            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            //hipFree(dev_odata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {


            //int blockSize = testBlockSize; // TODO optimize
            int blockSize = 512; 
            //512 seems to give best performance out of the values tested, but then scan on its own seems best at 32, so using 32 for scan and 512 for rest--seems to give better performance than same size for all the steps
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            int* dev_idata;
            int* dev_odata;
            int* dev_boolArray;
            int* dev_indices;
            //int* dev_
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));

            int scanBlockSize = 32;
            int dTarget = ilog2ceil(n);
            int pow2Size = 1 << dTarget;
            int nCap = pow2Size >> 1;
            dim3 fullBlocksPerGridScan((nCap + scanBlockSize - 1) / scanBlockSize);

            hipMalloc((void**)&dev_boolArray, pow2Size * sizeof(int));
            hipMalloc((void**)&dev_indices, pow2Size * sizeof(int));

            //hipMemset(dev_idata, 0, sizeof(int));
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemset(dev_odata, 0, sizeof(int) * n);
            hipMemset(dev_boolArray + n, 0, sizeof(int) * (pow2Size - n));
            




            timer().startGpuTimer();
            // TODO

            // map
            Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize >>>(n, dev_boolArray, dev_idata);

            // scan
            hipMemcpy(dev_indices, dev_boolArray, n * sizeof(int), hipMemcpyDeviceToDevice);
            // up-sweep
            int dExpo = 1; // = 2^(d)

            for (int d = 0; d < dTarget; ++d) {

                kernUpSweep << <fullBlocksPerGridScan, scanBlockSize >> > (nCap, dExpo, dev_indices);

                if (d < dTarget - 1) {
                    dExpo <<= 1;
                    nCap >>= 1;
                    fullBlocksPerGridScan.x = ((nCap + scanBlockSize - 1) / scanBlockSize); // Not sure this is totally the best way to set this but does massively reduce runtime (e.g. ~6ms to ~2ms)
                }

            }

            // down-sweep
            hipMemset(dev_indices + (pow2Size - 1), 0, sizeof(int));
            for (int d = dTarget - 1; d >= 0; --d) {
                kernDownSweep << <fullBlocksPerGridScan, scanBlockSize >> > (nCap, dExpo, dev_indices);
                if (d > 0) {
                    dExpo >>= 1;
                    nCap <<= 1;
                    fullBlocksPerGridScan.x = ((nCap + scanBlockSize - 1) / scanBlockSize);
                }
            }




            // scatter
            Common::kernScatter<<<fullBlocksPerGrid, blockSize >>>(n, dev_odata, dev_idata, dev_boolArray, dev_indices);

            


            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            int count;
            int countStep;
            hipMemcpy(&count, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&countStep, dev_boolArray + n - 1, sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_boolArray);
            hipFree(dev_indices);

            
            return count + countStep;
        }
    }
}
