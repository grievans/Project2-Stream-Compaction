#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        __global__ void kernUpSweep(int n, int dExpo, int* data/*, const int* idata*/) {
            // TODO I assume should invoke these with only the number of blocks actually used rather than with all the blocks when most don't do any, but will do following structure more literally first?
            // TODO probably worth trying out shared memory way--I guess would move loop into here w/ syncs then change external loop to just when needing to cross boundaries
            int d2 = dExpo << 1;
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return; // TODO pass in that cap?
            }
            int k = (index) * d2;
            //if (k >= n) {
                //return; // sounds like this way is generally better (more explicit that thread can stop)
            //}
            //if (k < n) {
                //odata[k + d2 - 1] = idata[k + dExpo - 1] + idata[k + d2 - 1];
                // does this need a separate idata odata? I think no others operate on it this step
                data[k + d2 - 1] += data[k + dExpo - 1];
            //}
        }
        __global__ void kernDownSweep(int n, int dExpo, int* data) {
            int d2 = dExpo << 1;
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return; // TODO pass in that cap?
            }
            int k = (index)*d2;
            //if (k >= n) {
                //return;
            //}
            int t = data[k + dExpo - 1];
            data[k + dExpo - 1] = data[k + d2 - 1];
            data[k + d2 - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO not finished yet, doesn't work yet
            int blockSize = 128; // TODO optimize
            int pow2Size = 1 << ilog2ceil(n);
            dim3 fullBlocksPerGrid((pow2Size >> 1 + blockSize - 1) / blockSize);
            // TODO I'm still not sure if getting fullBlocksPerGrid right 100%, might be overshooting
            int* dev_idata;
            int* dev_odata;

            // TODO note this pads to the whole next power of 2, was mentioned but can't recall if they said a way about that?
            // TODO I think want to rewrite into using shared memory way but that's extra credit so I think don't need to


            hipMalloc((void**)&dev_idata, pow2Size * sizeof(int));
            //hipMalloc((void**)&dev_odata, n * sizeof(int));
            
            hipMemset(dev_idata + n, 0, sizeof(int) * (pow2Size - n));
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice); 

            //hipMemset(dev_idata, 0, sizeof(int));
            //hipMemcpy(dev_idata + 1, idata, sizeof(int) * (n - 1), hipMemcpyHostToDevice);

            // TODO make sure to deal with size not 2^x

            timer().startGpuTimer();

            // TODO
            int dTarget = ilog2ceil(n); // TODO should these be out of timer?
            // up-sweep
            int dExpo = 1; // = 2^(d)
            int nCap = pow2Size >> 1;
            for (int d = 0; d < dTarget; ++d) {
                
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>> (nCap, dExpo, dev_idata);

                if (d < dTarget - 1) {
                    //fullBlocksPerGrid.x >>= 1;
                    //fullBlocksPerGrid = dim3((n / dExpo + blockSize - 1) / blockSize);
                    //fullBlocksPerGrid.x = (fullBlocksPerGrid.x - 1) / 2 + 1; // TODO reduce # of blocks accordingly
                    dExpo <<= 1;
                    nCap >>= 1;
                    fullBlocksPerGrid.x = ((nCap + blockSize - 1) / blockSize); // Not sure this is totally the best way to set this but does massively reduce runtime (e.g. ~6ms to ~2ms)
                    //std::swap(dev_idata, dev_odata);
                }
            }

            // down-sweep
            hipMemset(dev_idata + (pow2Size - 1), 0, sizeof(int)); // TODO make sure that's right
            //fullBlocksPerGrid = dim3((pow2Size + blockSize - 1) / blockSize);
            // TODO make sure dExpo right
            for (int d = dTarget - 1; d >= 0; --d) {
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>>(nCap, dExpo, dev_idata);
                // TODO make fullBlocksPerGrid right;
                //fullBlocksPerGrid.x = (dExpo + blockSize - 1) / blockSize; // TODO make sure set properly but works on super basic case
                dExpo >>= 1;
                nCap <<= 1;
                fullBlocksPerGrid.x = ((nCap + blockSize - 1) / blockSize);
            }


            timer().endGpuTimer();
            hipMemcpy(odata, dev_idata, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(dev_idata);
            //hipFree(dev_odata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
